#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <random>
#include <fstream>

//define the data set size for a cubic volume
#define DATAXSIZE 256
#define DATAYSIZE 256
#define DATAZSIZE 256

//define the chunk sizes that each threadblock will work on
#define BLKXSIZE 16
#define BLKYSIZE 4
#define BLKZSIZE 4

using namespace std;


__device__ double Laplacian(const double c[][DATAYSIZE][DATAXSIZE],
                            double dx, double dy, double dz, int x, int y, int z)
{
  int xp, xn, yp, yn, zp, zn;

  int nx = (int)DATAXSIZE - 1;
  int ny = (int)DATAYSIZE - 1;
  int nz = (int)DATAZSIZE - 1;

  xp = x+1;
  xn = x-1;
  yp = y+1;
  yn = y-1;
  zp = z+1;
  zn = z-1;

  if (xp > nx) xp = 0;
  if (yp > ny) yp = 0;
  if (zp > nz) zp = 0;
  if (xn < 0)  xn = nx;
  if (yn < 0)  yn = ny;
  if (zn < 0)  zn = nz;

  double cxx = (c[z][y][xp] + c[z][y][xn] - 2.0*c[z][y][x]) / (dx*dx);
  double cyy = (c[z][yp][x] + c[z][yn][x] - 2.0*c[z][y][x]) / (dy*dy);
  double czz = (c[zp][y][x] + c[zn][y][x] - 2.0*c[z][y][x]) / (dz*dz);

  double result = cxx + cyy + czz;

  return result;
}

__device__ double GradientX(const double phi[][DATAYSIZE][DATAXSIZE], 
                            double dx, double dy, double dz, int x, int y, int z)
{
  int nx = (int)DATAXSIZE - 1;
  int xp = x+1;
  int xn = x-1;

  if (xp > nx) xp = 0;
  if (xn < 0)  xn = nx;

  double phix = (phi[z][y][xp] - phi[z][y][xn]) / (2.0*dx);

  return phix;
}

__device__ double GradientY(const double phi[][DATAYSIZE][DATAXSIZE], 
                            double dx, double dy, double dz, int x, int y, int z)
{
  int ny = (int)DATAYSIZE - 1;
  int yp = y+1;
  int yn = y-1;

  if (yp > ny) yp = 0;
  if (yn < 0)  yn = ny;

  double phiy = (phi[z][yp][x] - phi[z][yn][x]) / (2.0*dy);

  return phiy;
}

__device__ double GradientZ(const double phi[][DATAYSIZE][DATAXSIZE],
                            double dx, double dy, double dz, int x, int y, int z)
{
  int nz = (int)DATAZSIZE - 1;
  int zp = z+1;
  int zn = z-1;

  if (zp > nz) zp = 0;
  if (zn < 0)  zn = nz;

  double phiz = (phi[zp][y][x] - phi[zn][y][x]) / (2.0*dz);

  return phiz;
}

__global__ void chemicalPotential(
    const double c[][DATAYSIZE][DATAXSIZE], 
    double mu[][DATAYSIZE][DATAXSIZE], 
    double dx,
    double dy,
    double dz,
    double gamma,
    double e_AA,
    double e_BB,
    double e_AB)
{

  unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned idy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned idz = blockIdx.z*blockDim.z + threadIdx.z;

  if ((idx < DATAXSIZE) && (idy < DATAYSIZE) && (idz < DATAZSIZE)) {

    mu[idz][idy][idx] = 4.5 * ( ( c[idz][idy][idx] + 1.0 ) * e_AA + 
        ( c[idz][idy][idx] - 1 ) * e_BB - 2.0 * c[idz][idy][idx] * e_AB ) + 
      3.0 * c[idz][idy][idx] + c[idz][idy][idx] * c[idz][idy][idx] * c[idz][idy][idx] - 
      gamma * Laplacian(c,dx,dy,dz,idx,idy,idz);
  }

}

__device__ double freeEnergy(double c, double e_AA, double e_BB, double e_AB)
{

  return (((9.0 / 4.0) * ((c*c+2.0*c+1.0)*e_AA+(c*c-2.0*c+1.0)*e_BB+
          2.0*(1.0-c*c)*e_AB)) + ((3.0/2.0) * c * c) + ((3.0/12.0) * c * c * c * c));

}

__global__ void localFreeEnergyFunctional(
    const double c[][DATAYSIZE][DATAXSIZE],
    double f[][DATAYSIZE][DATAXSIZE], 
    double dx,
    double dy,
    double dz,
    double gamma,
    double e_AA,
    double e_BB,
    double e_AB)
{

  unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned idy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned idz = blockIdx.z*blockDim.z + threadIdx.z;

  if ((idx < DATAXSIZE) && (idy < DATAYSIZE) && (idz < DATAZSIZE)) {

    f[idz][idy][idx] = freeEnergy(c[idz][idy][idx],e_AA,e_BB,e_AB) + (gamma / 2.0) * (
        GradientX(c,dx,dy,dz,idx,idy,idz) * GradientX(c,dx,dy,dz,idx,idy,idz) + 
        GradientY(c,dx,dy,dz,idx,idy,idz) * GradientY(c,dx,dy,dz,idx,idy,idz) + 
        GradientZ(c,dx,dy,dz,idx,idy,idz) * GradientZ(c,dx,dy,dz,idx,idy,idz));
  }

}

__global__ void cahnHilliard(
    double cnew[][DATAYSIZE][DATAXSIZE], 
    const double cold[][DATAYSIZE][DATAXSIZE], 
    const double mu[][DATAYSIZE][DATAXSIZE],
    double D,
    double dt,
    double dx,
    double dy,
    double dz)
{
  unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned idy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned idz = blockIdx.z*blockDim.z + threadIdx.z;
  if ((idx < DATAXSIZE) && (idy < DATAYSIZE) && (idz < DATAZSIZE)) {
    cnew[idz][idy][idx] = cold[idz][idy][idx] + dt * D * Laplacian(mu,dx,dy,dz,idx,idy,idz);
  }
}

__global__ void Swap(double cnew[][DATAYSIZE][DATAXSIZE], double cold[][DATAYSIZE][DATAXSIZE])
{
  unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
  unsigned idy = blockIdx.y*blockDim.y + threadIdx.y;
  unsigned idz = blockIdx.z*blockDim.z + threadIdx.z;
  double tmp;    

  if ((idx < DATAXSIZE) && (idy < DATAYSIZE) && (idz < DATAZSIZE)) {
    tmp = cnew[idz][idy][idx];
    cnew[idz][idy][idx] = cold[idz][idy][idx];
    cold[idz][idy][idx] = tmp;
  }

}

void initialization(double c[][DATAYSIZE][DATAXSIZE])
{
  srand(2);
  for (unsigned int idz = 0.0; idz < DATAZSIZE; idz++) {
    for (unsigned int idy = 0.0; idy < DATAYSIZE; idy++) {
      for (unsigned int idx = 0.0; idx < DATAXSIZE; idx++) {
        double f = (double)rand() / RAND_MAX;
        c[idz][idy][idx] = -1.0 + 2.0*f;
      }
    }
  }
}

double integral(const double c[][DATAYSIZE][DATAXSIZE], int nx, int ny, int nz)
{
  double summation = 0.0;  

  for (int k = 0; k < nz; k++)
    for(int j = 0; j < ny; j++)
      for(int i = 0; i < nx; i++)
        summation = summation + c[k][j][i];

  return summation;
}

int main(int argc, char *argv[])
{
  const double dx = 1.0;
  const double dy = 1.0;
  const double dz = 1.0;
  const double dt = 0.01;
  const double e_AA = -(2.0/9.0);
  const double e_BB = -(2.0/9.0);
  const double e_AB = (2.0/9.0);
  const int t_f = atoi(argv[1]);    // default value: 25000
#ifndef DEBUG
  const int t_freq = t_f; 
#else
  const int t_freq = 10;
#endif
  const double gamma = 0.5;
  const double D = 1.0;

  string name_c = "./out/integral_c.txt";
  ofstream ofile_c (name_c);

  string name_mu = "./out/integral_mu.txt";
  ofstream ofile_mu (name_mu);

  string name_f = "./out/integral_f.txt";
  ofstream ofile_f (name_f);

  typedef double nRarray[DATAYSIZE][DATAXSIZE];

  // overall data set sizes
  const int nx = DATAXSIZE;
  const int ny = DATAYSIZE;
  const int nz = DATAZSIZE;
  const int vol = nx * ny * nz;

  // pointers for data set storage via malloc
  nRarray *c_host; // storage for result stored on host
  nRarray *mu_host;
  nRarray *f_host;
  nRarray *d_cold; // storage for result computed on device
  nRarray *d_cnew;
  nRarray *d_muold;
  nRarray *d_fold;

  if ((c_host = (nRarray *)malloc(vol*sizeof(double))) == 0) {
    fprintf(stderr,"c_host malloc failed\n"); 
    return 1;
  }
  if ((mu_host = (nRarray *)malloc(vol*sizeof(double))) == 0) {
    fprintf(stderr,"mu_host malloc failed\n"); 
    return 1;
  }
  if ((f_host = (nRarray *)malloc(vol*sizeof(double))) == 0) {
    fprintf(stderr,"f_host malloc failed\n"); 
    return 1;
  }

  hipMalloc((void **) &d_cold, vol*sizeof(double));
  hipMalloc((void **) &d_cnew, vol*sizeof(double));
  hipMalloc((void **) &d_muold, vol*sizeof(double));
  hipMalloc((void **) &d_fold, vol*sizeof(double));

  initialization(c_host);

  double integral_c = 0.0;
  double integral_mu = 0.0;
  double integral_f = 0.0;


  hipMemcpy(d_cold, c_host, (vol*sizeof(double)), hipMemcpyHostToDevice);

  double clock_d = double(clock()) / CLOCKS_PER_SEC;


  const dim3 blockSize(BLKXSIZE, BLKYSIZE, BLKZSIZE);
  const dim3 gridSize((DATAXSIZE+BLKXSIZE-1)/BLKXSIZE, 
                      (DATAYSIZE+BLKYSIZE-1)/BLKYSIZE,
                      (DATAZSIZE+BLKZSIZE-1)/BLKZSIZE);

  for (int t = 0; t < t_f; t++) {

    chemicalPotential<<<gridSize, blockSize>>>(d_cold,d_muold,dx,dy,dz,gamma,e_AA,e_BB,e_AB);
    localFreeEnergyFunctional<<<gridSize, blockSize>>>(d_cold,d_fold,dx,dy,dz,gamma,e_AA,e_BB,e_AB);
    cahnHilliard<<<gridSize, blockSize>>>(d_cnew,d_cold,d_muold,D,dt,dx,dy,dz);

    if (t > 0 && t % (t_freq - 1) == 0) {
      hipMemcpy(c_host, d_cnew, (vol*sizeof(double)), hipMemcpyDeviceToHost);

      hipMemcpy(mu_host, d_muold, (vol*sizeof(double)), hipMemcpyDeviceToHost);

      hipMemcpy(f_host, d_fold, (vol*sizeof(double)), hipMemcpyDeviceToHost);

      integral_c = integral(c_host,nx,ny,nz);

      ofile_c << t << "," << integral_c << endl;

      integral_mu = integral(mu_host,nx,ny,nz);

      ofile_mu << t << "," << integral_mu << endl;

      integral_f = integral(f_host,nx,ny,nz);

      ofile_f << t << "," << integral_f << endl;
    }

    Swap<<<gridSize, blockSize>>>(d_cnew, d_cold);
  }

  hipDeviceSynchronize();
  clock_d = double(clock()) / CLOCKS_PER_SEC - clock_d; 
  printf("Exeuction time on the GPU (%d iterations) = %.3fms\n", t_f, clock_d*1e3);

  free(c_host);
  free(mu_host);
  free(f_host);
  hipFree(d_cold);
  hipFree(d_cnew);
  hipFree(d_muold);
  hipFree(d_fold);
  return 0;
}
