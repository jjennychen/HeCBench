#include <assert.h>
#include <stdlib.h>
#include <unistd.h>
#include <chrono>
#include <iostream>
#include <type_traits> // is_same
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

using namespace std;

template <typename T>
void gemmBatched(
  int lower,
  int upper,
  int num,
  int reps,
  int verbose) 
{
  if(verbose) cout << "initializing inputs" << endl;
  size_t matrices_size = upper * upper * num * sizeof(T);
  size_t vectors_size = upper * num * sizeof(T);

  T *matrices = (T*)malloc(matrices_size);
  assert(matrices);

  T *vectors = (T*)malloc(vectors_size);
  assert(vectors);

  srand48(48);
  for(int i = 0; i < num * upper * upper; i++)
    matrices[i] = static_cast<T>(drand48());

  for(int i = 0; i < num * upper; i++)
    vectors[i] = static_cast<T>(drand48());

  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;

  stat = hipblasCreate(&handle);
  if(stat != HIPBLAS_STATUS_SUCCESS){
    cerr << "cublas init failed" << endl;
    exit(1);
  }

  if(verbose) cout << "allocating device variables" << endl;

  // allocate input space on device
  T *devMatrices;
  cudaStat = hipMalloc((void**)&devMatrices, matrices_size);
  assert(!cudaStat);

  T *devVectors;
  cudaStat = hipMalloc((void**)&devVectors, vectors_size);
  assert(!cudaStat);

  // allocate result space on device
  T *devResult;
  cudaStat = hipMalloc((void**)&devResult, vectors_size);

  assert(!cudaStat);

  if(verbose) cout << "copying data to device" << endl;
  // copy data to device
  cudaStat = 
    hipMemcpy(devMatrices, matrices, matrices_size, hipMemcpyHostToDevice);

  assert(!cudaStat);
  
  cudaStat = 
    hipMemcpy(devVectors, vectors, vectors_size, hipMemcpyHostToDevice);

  assert(!cudaStat);

  // create lists of device pointers to inputs and outputs
  T **AList = 0, **BList = 0, **CList = 0;

  AList = (T**)malloc(num * sizeof(T*));
  BList = (T**)malloc(num * sizeof(T*));
  CList = (T**)malloc(num * sizeof(T*));

  int lda = upper, // lda >= max(1,m)
      ldb = upper, // ldb >= max(1,k)
      ldc = upper; // ldc >= max(1,m)

  const T alpha = 1.0f, beta = 0.0f;
  for(int i = 0; i < num; i++){
    // each array of dim. lda x k
    AList[i] = devMatrices + upper * upper * i;
    // each array of dim. ldb x n
    BList[i] = devVectors + upper * i;
    // each array of dim. ldc x n
    CList[i] = devResult + upper * i;
  }

  // copy pointer lists to device
  T **devAList, **devBList, **devCList;
  cudaStat = hipMalloc((void**)&devAList, num * sizeof(T*));
  assert(!cudaStat);

  cudaStat = hipMalloc((void**)&devBList, num * sizeof(T*));
  assert(!cudaStat);

  cudaStat = hipMalloc((void**)&devCList, num * sizeof(T*));
  assert(!cudaStat);

  cudaStat = hipMemcpy(devAList, AList, num * sizeof(T*), hipMemcpyHostToDevice);
  assert(!cudaStat);
  
  cudaStat = hipMemcpy(devBList, BList, num * sizeof(T*), hipMemcpyHostToDevice);
  assert(!cudaStat);

  cudaStat = hipMemcpy(devCList, CList, num * sizeof(T*), hipMemcpyHostToDevice);
  assert(!cudaStat);


  /* perform <num> <size x size> x <size x 1> multiplications 
     with distinct matrices
   */
#define GEMM_BATCHED_PARAMETERS handle,              \
                                HIPBLAS_OP_N,         \
                                HIPBLAS_OP_N,         \
                                m, n, k,             \
                                &alpha,              \
                                (const T**)devAList, \
                                lda,                 \
                                (const T**)devBList, \
                                ldb,                 \
                                &beta,               \
                                devCList,            \
                                ldc,                 \
                                num

  for(int size = lower; size <= upper; size++){
    if(verbose) cout << "running with <size x size> x <size x 1> " << size << endl;
    double sum = 0.0;
    const int m = size, n = 1, k = size;
    for(int rep = 0; rep <= reps; rep++){
      auto start = std::chrono::steady_clock::now();

      if constexpr (std::is_same_v<T, double>)
        stat = hipblasDgemmBatched(GEMM_BATCHED_PARAMETERS);
      else if constexpr (std::is_same_v<T, float>)
        stat = hipblasSgemmBatched(GEMM_BATCHED_PARAMETERS);
      else if constexpr (std::is_same_v<T, __half>)
        stat = hipblasHgemmBatched(GEMM_BATCHED_PARAMETERS);

      hipDeviceSynchronize();
      auto end = std::chrono::steady_clock::now();
      auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
      auto elapsed = time * 1e-3;

      if(stat != HIPBLAS_STATUS_SUCCESS){
        cerr << "cublasXgemmBatched failed" << endl;
        break;
      }

      if (rep != 0) sum += elapsed;
      
      if(verbose)
	cout << "size " << size << ": " << elapsed << " us; " 
	     << elapsed / num << " us per operation" << endl;
    }
    cout << "size " << size << " average execution time: " << sum/reps << " us; "
	 << sum / reps / num << " us per operation" << endl;
  }

  hipFree(devMatrices);
  hipFree(devVectors);
  hipFree(devResult);
  hipFree(devAList);
  hipFree(devBList);
  hipFree(devCList);

  free(matrices);
  free(vectors);
  free(AList);
  free(BList);
  free(CList);
}

int main(int argc, char ** argv){

  int status;
  int lower = 2;    // lower bound to the matrix dimension
  int upper = 100;  // upper bound to the matrix dimension
  int num = 25000;  // batch size
  int reps = 10;
  int verbose = 0;
  
  while((status = getopt(argc, argv, "l:u:n:r:v")) != -1){
    switch(status){
    case 'l':
      lower = strtoul(optarg, 0, 0);
      break;
    case 'u':
      upper = strtoul(optarg, 0, 0);
      break;
    case 'n':
      num = strtoul(optarg, 0, 0);  // batch size
      break;
    case 'r':
      reps = strtoul(optarg, 0, 0);
      break;
    case 'v':
      verbose = 1;
      break;
    default:
      cerr << "invalid argument: " << status << endl;
      exit(1);
    }
  }

  cout << "running with" << " lower: " << lower << " upper: " << upper
       << " num: " << num << " reps: " << reps << endl;

  cout << ">>>>>>>>>>>>>>> Half precision gemmBatched >>>>>>>>>>>>>>> " << endl;
  gemmBatched<__half>(lower, upper, num, reps, verbose);
  cout << ">>>>>>>>>>>>>>> Single precision gemmBatched >>>>>>>>>>>>>>> " << endl;
  gemmBatched<float>(lower, upper, num, reps, verbose);
  cout << ">>>>>>>>>>>>>>> Double precision gemmBatched >>>>>>>>>>>>>>> " << endl;
  gemmBatched<double>(lower, upper, num, reps, verbose);
      
  return 0;
}